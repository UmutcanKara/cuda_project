
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <limits.h>
#include <hiprand/hiprand_kernel.h>

#define TRUE 1
#define FALSE 0
#define INFNTY INT_MAX

typedef int boolean;



__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &state[id]); // Initialize state
}


/* Generates a random undirected graph represented by an adjacency matrix */
__global__ void generate_random_graph_kernel(int V, int *adjacency_matrix, hiprandState *state, int *randomNumbers)
{    
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
    int stride = blockDim.x*gridDim.x;

    for (int i = tid; i < V; i+=stride)
    {
        for (int j = 0; j < V; j++)
        {
            int r;
            hiprandState localState = state[i];
            r = (int)(hiprand_uniform(&localState) * 10.0f);

            if (i < j) {
                adjacency_matrix[i * V + j] = r;                 /* Assign a random value corresponding to the edge */
                adjacency_matrix[j * V + i] = r; /* Graph is undirected, the adjacency matrix is symmetric */
               }
            state[i] = localState;
            }
        adjacency_matrix[i * V + i] = 0;
        
        }
    }


__global__ void print_adjacency_matrix_kernel(int V, int *adjacency_matrix)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for(int i=tid; i < V; i+=stride)
        {
            printf("matrix[%d] = %d\n",i,adjacency_matrix[i]);
        }
}


__global__ void dijkstra_kernel(int V, int *adjacency_matrix, int *len, int *temp_distance, boolean *visited)
{

    /* Computing the All Pairs Shortest Paths (APSP) in the graph */
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int source = tid; source < V; source+=stride)
        {
        for (int i = 0; i < V; i++) /* Initialize vars arrays to current source */
        {
            visited[i] = FALSE;
            temp_distance[i] = INFNTY;
            len[source * V + i] = INFNTY;
        }

        len[source * V + source] = 0; /* Set the distance of the source vertex as 0 */

        for (int count = 0; count < V - 1; count++)
        {
            /* Finds the vertex with the minimum distance from the current source vertex */
            int min_distance = INFNTY; /* Init value */
            int min_index = -1;
        
            for (int v = 0; v < V; v++) /* Iterates over all vertices */
            {
                if (!visited[v] && len[v] <= min_distance)
                {
                    min_distance = len[v];
                    min_index = v;
                }
            }
            
            int current_vertex = min_index;
            visited[current_vertex] = TRUE;

            for (int v = 0; v < V; v++)
            {
                int weight = adjacency_matrix[current_vertex * V + v];
                if (!visited[v] && weight && len[source * V + current_vertex] != INFNTY &&
                    len[source * V + current_vertex] + weight < len[source * V + v])
                {
                    /* Updating the distance is beneficial */
                    len[source * V + v] = len[source * V + current_vertex] + weight;
                    temp_distance[v] = len[source * V + v];
                }
            }
        }
        }
}



int main(int argc, char **argv)
{

    if (argc != 2)
    {
        printf("USAGE: ./dijkstra_parallel <number_of_vertices>\n");
        return 1;
    }
    
    int deviceId;
    hipGetDevice(&deviceId);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, deviceId);
    int sm_count = props.multiProcessorCount;
    int warp_size = props.warpSize;
    
    int threadsPerBlock = warp_size*16;
    int numberOfBlocks = sm_count*16;

    int V = atoi(argv[1]); /* Number of vertices */
    int * adjacency_matrix;
    int *len, *temp_distance;
    boolean *visited;
    
    hipMalloc(&visited, V * sizeof(boolean));
    hipMalloc(&len, V * V * sizeof(int));
    hipMalloc(&adjacency_matrix, V * V * sizeof(int));
    hipMalloc(&temp_distance, V * sizeof(boolean));
    
    hiprandState *d_states;
    int* d_randomNumbers;
    hipMalloc(&d_randomNumbers, V * V * sizeof(int));
    hipMalloc(&d_states, V * V * sizeof(hiprandState));

    // Setup CURAND states
    setup_kernel<<<numberOfBlocks, threadsPerBlock>>>(d_states, time(NULL));
    
    clock_t start = clock(); /* Records the start time for measuring the execution time */
    generate_random_graph_kernel<<<numberOfBlocks, threadsPerBlock>>>(V, adjacency_matrix, d_states, d_randomNumbers);
    hipDeviceSynchronize();
    clock_t end = clock();   /* Records the end time for measuring the execution time */
    float seconds = (float)(end - start) / CLOCKS_PER_SEC;
    printf("TIME TO CREATE GRAPH ON GPU = %f SECS\n", seconds);
    
    start = clock(); /* Records the start time for measuring the execution time */

    dijkstra_kernel<<<numberOfBlocks, threadsPerBlock>>>(V, adjacency_matrix, len, temp_distance, visited);
    hipDeviceSynchronize();
    end = clock(); /* Records the end time for measuring the execution time */
    seconds = (float)(end - start) / CLOCKS_PER_SEC;
    printf("TIME FOR ALL PAIRS DIJKSTRA ON GPU = %f SECS\n", seconds);
    
    hipFree(visited);
    hipFree(len);
    hipFree(temp_distance);
    hipFree(adjacency_matrix);
    
    return 0;
}
